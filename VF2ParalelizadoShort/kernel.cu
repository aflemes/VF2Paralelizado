#include "hip/hip_runtime.h"
const static int maxv = 40;
const static int maxe = 80;
const int MAX_GRAPHS_DB = 8192;
const int MAX_GRAPHS_QUERY = 2;
int NBLOCKS = 1, NTHREADS = 1;
const int maxThreadsPerBlock = 256, minBlocksPerMultiprocessor = 8;
const int MAX = 512;
__device__
int controle[MAX];

#include "head.h"
#include "class.h"
#include "signature.h"

#define swap(A,B) { float temp = A; A = B; B = temp;}

const char *QueryPath[MAX_GRAPHS_QUERY]; // Query file path vector
int QueryPathPointer[MAX_GRAPHS_QUERY];
int DBGraphSize, QueryGraphSize, QueryPathSize;

Graph DBGraph[MAX_GRAPHS_DB], QueryGraph[MAX_GRAPHS_QUERY], *vec;

unsigned int matches[MAX_GRAPHS_QUERY];

char *queryPath, *dbPath;

void init()
{
	string qry = "data/query/Q4.min.my";
	string db = "data/db/Q8192.data";
	//string db = "data/db/mygraphdb.min.data";

	
	if (queryPath == NULL) {		
		queryPath = (char*)malloc(size(qry) + 1 * sizeof(char));
		strcpy_s(queryPath, size(qry) + 1, qry.c_str());
	}
	
	if (dbPath == NULL) {
		dbPath = (char*)malloc(size(db) + 1 * sizeof(char));
		strcpy_s(dbPath, size(db) + 1, db.c_str());		
	}
	memset(matches, 0, MAX_GRAPHS_QUERY * sizeof(int));

}

void input()
{	
	ReadQuery(queryPath);
	//le o(s) grafo(s) modelo(s)
	ReadDB(dbPath);
	puts("Read Data Finished!");
}

void ReadFile(char *path, int &graphSize, int MAX_GRAPHS)
{
	bool eof = false;
	graphSize = 0;

	ifstream fin;

	fin.open(path);

	if (!fin.is_open()) {
		printf("Arquivo %s nao encontrado \n", path);
		return;
	}

	vec = (Graph*)malloc(MAX_GRAPHS * sizeof(Graph));
	vec[graphSize].aloca();

	string buff;
	int n = -1;
	int m, l;
	int p, q;
	while (getline(fin, buff))
	{

		if (buff.size() == 0) continue;
		if (buff == "t # -1")
		{
			eof = true;
			graphSize++;			
			break;
		}
		if (buff[0] == 't')
		{
			n++;
			if (n == 0) continue;

			graphSize++;
			vec[graphSize].aloca();
		}
		else if (buff[0] == 'v')
		{
			sscanf_s(buff.c_str(), "v %d %d", &m, &l);
			vec[graphSize].addv(m, l);
		}
		else if (buff[0] == 'e')
		{
			sscanf_s(buff.c_str(), "e %d %d %d", &p, &q, &l);
			vec[graphSize].adde(p, q, l);

		}
		else puts("Error!");
	}

	if (!eof)
		printf("Nao foi encontrado o fim do arquivo (t #-1) \n");

	fin.close();
}

void ReadDB(char *path)
{
	ReadFile(path, DBGraphSize, MAX_GRAPHS_DB);
	
	for (int i = 0; i < DBGraphSize;i++) {
		DBGraph[i].en = vec[i].en;
		DBGraph[i].vn = vec[i].vn;

		DBGraph[i].vtx  = vec[i].vtx;
		DBGraph[i].edge = vec[i].edge;
		DBGraph[i].head = vec[i].head;
	}
}
void ReadQuery(char *path)
{
	printf("read query");
	ReadFile(path, QueryGraphSize, MAX_GRAPHS_QUERY);

	for (int i = 0; i < QueryGraphSize;i++) {
		QueryGraph[i].en = vec[i].en;
		QueryGraph[i].vn = vec[i].vn;

		QueryGraph[i].vtx  = vec[i].vtx;
		QueryGraph[i].edge = vec[i].edge;
		QueryGraph[i].head = vec[i].head;
	}

}

__device__
void initGraph(Graph &src, Graph &dest) {
	dest.en = src.en;
	dest.vn = src.vn;

	for (int k = 0; k < src.en;k++) {
		dest.edge[k] = src.edge[k];
		dest.head[k] = src.head[k];
	}

	for (int k = 0; k < src.vn;k++) {
		dest.vtx[k] = src.vtx[k];
	}
}
__device__
void GenRevGraph(const Graph &src, Graph &dst)
{
	for (int i = 0; i < src.vn; i++)
		dst.addv(src.vtx[i].id, src.vtx[i].label);

	for (int i = 0; i < src.en; i++)
		dst.addse(src.edge[i].v, src.edge[i].u, src.edge[i].label);
}

__device__
void printGraph(Graph grafo[], int size) {
	for (int i = 0;i < size; i++) {
		printf("Indice %d Graph[i].en %d Graph[i].vn %d => \n", i, grafo[i].en, grafo[i].vn);

		for (int j = 0; j < grafo[i].en;j++) {
			printf("indice %d Edge[j].u %d Edge[j].v %d Edge[j].next %d\n", j, grafo[i].edge[j].u, grafo[i].edge[j].v, grafo[i].edge[j].next);
		}
		for (int j = 0; j < grafo[i].vn;j++) {
			printf("indice %d Vtx[j].id %d Vtx[j].label %d \n", j, grafo[i].vtx[j].id, grafo[i].vtx[j].label);
		}
	}
}
__device__
void ClearArrays(VetAuxiliares &vetAux) {
	for (int i = 0; i < maxv;i++) {
		vetAux.m1[i] = 0, vetAux.m2[i] = 0;
		vetAux.tin1[i] = 0, vetAux.tin2[i] = 0;
		vetAux.tout1[i] = 0, vetAux.tout2[i] = 0;
		vetAux.n1[i] = 0, vetAux.n2[i] = 0;
		vetAux.ns1[i] = 0, vetAux.ns2[i] = 0;
		vetAux.t1[i] = 0, vetAux.t2[i] = 0;
	}

	vetAux.sizeM1 = 0, vetAux.sizeM2 = 0;
	vetAux.sizeTin1 = 0, vetAux.sizeTin2 = 0;
	vetAux.sizeTout1 = 0, vetAux.sizeTout2 = 0;
	vetAux.sizeN1 = 0, vetAux.sizeN2 = 0;
	vetAux.sizeNS1 = 0, vetAux.sizeNS2 = 0;
}

__device__
void quicksort(int ls[], int l, int r) {
	int i, j, k, p, q;
	int v;
	if (r <= l)
		return;
	v = ls[r];
	i = l - 1;
	j = r;
	p = l - 1;
	q = r;
	for (;;) {
		while (ls[++i] < v);
		while (v < ls[--j])
			if (j == l)
				break;
		if (i >= j)
			break;
		swap(ls[i], ls[j]);
		if (ls[i] == v) {
			p++;
			swap(ls[p], ls[i]);
		}
		if (v == ls[j]) {
			q--;
			swap(ls[q], ls[j]);
		}
	}
	swap(ls[i], ls[r]);
	j = i - 1;
	i++;
	for (k = l; k < p; k++, j--)
		swap(ls[k], ls[j]);
	for (k = r - 1; k > q; k--, i++)
		swap(ls[k], ls[i]);

	quicksort(ls, l, j);
	quicksort(ls, i, r);
}
__device__
int Union(int arr1[], int arr2[], int arr3[], int m, int n)
{
	int i = 0, j = 0, x = 0;

	while (i < m && j < n) {
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
		}
		else
			if (arr2[j] < arr1[i]) {
				arr3[x++] = arr2[j++];
			}
			else {
				arr3[x++] = arr2[j++];
				i++;
			}
	}

	/* Print remaining elements of the larger array */
	while (i < m)
		arr3[x++] = arr1[i++];
	while (j < n)
		arr3[x++] = arr2[j++];

	return x;
}
__device__
int Difference(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, print smaller 
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
			k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++;
				k++;
			}
		// Skip common element 
			else {
				i++;
				j++;
			}
	}

	// printing remaining elements 
	while (i < n1) {

		arr3[x++] = arr1[i++];
		k++;
	}
	while (j < n2) {
		arr2[x++] = arr1[j++];
		k++;
	}

	return x;
}
__device__
int Intersection(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, jump
		if (arr1[i] < arr2[j]) {
			i++, k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++, k++;
			}
			else {
				arr3[x++] = arr1[i++];
				j++;
			}
	}

	return x;
}

Graph* alocaGraph(Graph *Grafo, int GraphSize) {
	Graph *GraphHost, *GraphCUDA;

	GraphHost = (Graph*)malloc(GraphSize * sizeof(Graph));

	for (int k = 0;k < GraphSize;k++) {
		Vertex *vtx;
		Edge *edge;
		int *head;

		if (hipMalloc((void **)&vtx, Grafo[k].vn * sizeof(Vertex)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&edge, Grafo[k].en * sizeof(Edge)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&head, maxe * sizeof(int)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar o head \n");
		}

		if (hipMemcpy(vtx, Grafo[k].vtx, Grafo[k].vn * sizeof(Vertex), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar os vertices \n");
		}

		if (hipMemcpy(edge, Grafo[k].edge, Grafo[k].en * sizeof(Edge), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar as arestas \n");
		}

		if (hipMemcpy(head, Grafo[k].head, maxe * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar o head \n");
		}

		GraphHost[k].vtx = vtx;
		GraphHost[k].edge = edge;
		GraphHost[k].head = head;
		GraphHost[k].en = Grafo[k].en;
		GraphHost[k].vn = Grafo[k].vn;
	}
	
	//int sizeofGrafo = GraphSize * (sizeof(Graph) + (maxv * sizeof(Vertex)) + (2 * maxe * sizeof(Edge)));
	//printf("GraphSize %d Graph mem. usage => %d \nGraph size %d \nVertex size => %d\nEdge size => %d \n", GraphSize, sizeofGrafo,sizeof(Graph), sizeof(Vertex), sizeof(Edge));

	hipMalloc((void **)&GraphCUDA, GraphSize * sizeof(Graph));
	hipMemcpy(GraphCUDA, GraphHost, (sizeof(Graph) * GraphSize), hipMemcpyHostToDevice);

	return GraphCUDA;
}


__device__
bool FinalCheck(const State &s, Graph &pat, Graph &g)
{
	for (int i = 0;i < pat.en;i++)
	{
		Edge e1 = pat.edge[i];
		bool flag = 0;

		for (int j = g.head[s.core1[e1.u]];~j;j = g.edge[j].next)
		{
			Edge e2 = g.edge[j];

			if (e1.label == e2.label&&s.core1[e1.v] == e2.v)
			{
				flag = 1;
				break;
			}
		}
		if (!flag) return 0;
	}
	return 1;
}
__device__
void CalDFSVec(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g)
{
	ClearArrays(vetAux);

	for (int i = 0; i < s.TAM;i++) {
		vetAux.m1[vetAux.sizeM1++] = s.first[i];
		vetAux.m2[vetAux.sizeM2++] = s.second[i];
	}

	if (vetAux.sizeM1 > 0 && vetAux.sizeM2 > 0) {
		quicksort(vetAux.m1, 0, vetAux.sizeM1 - 1);
		quicksort(vetAux.m2, 0, vetAux.sizeM2 - 1);
	}

	for (int i = 0; i < pat.vn; i++) {
		if (s.out1[i])
			vetAux.tout1[vetAux.sizeTout1++] = i;
		if (s.in1[i]) {
			vetAux.tin1[vetAux.sizeTin1++] = i;
		}
		vetAux.n1[vetAux.sizeN1++] = i;
	}

	for (int i = 0; i < g.vn; i++) {
		if (s.out2[i])
			vetAux.tout2[vetAux.sizeTout2++] = i;
		if (s.in2[i])
			vetAux.tin2[vetAux.sizeTin2++] = i;
		vetAux.n2[vetAux.sizeN2++] = i;
	}

	vetAux.sizeT1 = Union(vetAux.tin1, vetAux.tout1, vetAux.t1, vetAux.sizeTin1, vetAux.sizeTout1);

	vetAux.sizeT2 = Union(vetAux.tin1, vetAux.tout2, vetAux.t2, vetAux.sizeTin1, vetAux.sizeTout2);

	int tmp[maxv], sizeTmp;

	sizeTmp = Difference(vetAux.n1, vetAux.m1, tmp, vetAux.sizeN1, vetAux.sizeM1);

	vetAux.sizeNS1 = Difference(tmp, vetAux.t1, vetAux.ns1, sizeTmp, vetAux.sizeT1);

	sizeTmp = Difference(vetAux.n2, vetAux.m2, tmp, vetAux.sizeN2, vetAux.sizeM2);

	vetAux.sizeNS2 = Difference(tmp, vetAux.t2, vetAux.ns2, sizeTmp, vetAux.sizeT2);
}

__device__
bool check(const State &s, int a, int b, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Check vertex label
	if (pat.vtx[a].label != g.vtx[b].label) return 0;

	// Check edge label
	CalCheckVec(a, b, vetAux, pat, g, revpat, revg);

	// Feasibility
	if (CheckPrev(s, a, b, vetAux) && CheckSucc(s, a, b, vetAux) && CheckIn(vetAux) && CheckOut(vetAux) && CheckNew(vetAux)) return 1;

	return 0;
}

__device__
int GenPairs(const State &s, int allPairsFirst[], int allPairsSecond[], VetAuxiliares &vetAux, Graph &pat, Graph &g)
{
	int sizeAllPairs = 0;

	CalDFSVec(s, vetAux, pat, g);

	/*if (vetAux.sizeTout1 > 0 && vetAux.sizeTout2 > 0) {
		allPairsFirst = (int*)malloc(vetAux.sizeTout1 * vetAux.sizeTout2 * sizeof(int));
		allPairsSecond = (int*)malloc(vetAux.sizeTout1 * vetAux.sizeTout2 * sizeof(int));
	}*/

	for (int i = 0; i < (int)vetAux.sizeTout1; i++)
		for (int j = 0; j < (int)vetAux.sizeTout2; j++) {
			allPairsFirst[sizeAllPairs] = vetAux.tout1[i], allPairsSecond[sizeAllPairs++] = vetAux.tout2[j];
		}

	if (sizeAllPairs > 0)
	{
		return sizeAllPairs;
	}

	/*if (vetAux.sizeTin1 > 0 && vetAux.sizeTin2 > 0) {
		allPairsFirst = (int*)malloc(vetAux.sizeTin1 * vetAux.sizeTin2 * sizeof(int));
		allPairsSecond = (int*)malloc(vetAux.sizeTin1 * vetAux.sizeTin2 * sizeof(int));
	}*/

	for (int i = 0; i < (int)vetAux.sizeTin1; i++)
		for (int j = 0; j < (int)vetAux.sizeTin2; j++) {
			allPairsFirst[sizeAllPairs] = vetAux.tin1[i], allPairsSecond[sizeAllPairs++] = vetAux.tin2[j];
		}

	if (sizeAllPairs > 0)
	{
		return sizeAllPairs;
	}

	int temp1[maxv], temp2[maxv];
	int sizeTemp1 = 0, sizeTemp2 = 0;

	for (int i = 0; i < pat.vn; i++)
		if (s.core1[i] == -1)
			temp1[sizeTemp1++] = i;
	
	for (int i = 0; i < g.vn; i++)
		if (s.core2[i] == -1)
			temp2[sizeTemp2++] = i;

	/*allPairsFirst = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));
	allPairsSecond = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));*/

	for (int i = 0; i < sizeTemp1; i++)
		for (int j = 0; j < sizeTemp2; j++) {
			allPairsFirst[sizeAllPairs] = temp1[i], allPairsSecond[sizeAllPairs++] = temp2[j];
		}

	return sizeAllPairs;
}
__device__
int CheckPairs(const State &s, int allPairsFirst[], int allPairsSecond[], int candiPairsFirst[], int candiPairsSecond[], int sizeAllPairs, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	int sizeCandiPairs = 0;

	/*candiPairsFirst = (int*)malloc(sizeAllPairs * sizeof(int));
	candiPairsSecond = (int*)malloc(sizeAllPairs * sizeof(int));*/

	for (int i = 0; i < sizeAllPairs; i++) {
		if (check(s, allPairsFirst[i], allPairsSecond[i], vetAux, pat, g, revpat, revg)) {
			candiPairsFirst[sizeCandiPairs] = allPairsFirst[i];
			candiPairsSecond[sizeCandiPairs++] = allPairsSecond[i];
		}
	}

	return sizeCandiPairs;
}
__device__
void UpdateState(State &s, int a, int b, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Update core,in,out
	for (int i = 0; i < pat.vn; i++)
	{
		s.core1[a] = b;
		s.in1[a] = 0;
		s.out1[a] = 0;
	}
	for (int i = 0; i < g.vn; i++)
	{
		s.core2[b] = a;
		s.in2[b] = 0;
		s.out2[b] = 0;
	}

	for (int i = pat.head[a]; ~i; i = pat.edge[i].next)
	{
		int v = pat.edge[i].v;
		if (s.core1[v] == -1)
			s.out1[v] = 1;
	}
	// Add new in1
	for (int i = revpat.head[a]; ~i; i = revpat.edge[i].next)
	{
		int v = revpat.edge[i].v;
		if (s.core1[v] == -1)
			s.in1[v] = 1;
	}
	// Add new out2
	for (int i = g.head[b]; ~i; i = g.edge[i].next)
	{
		int v = g.edge[i].v;
		if (s.core2[v] == -1)
			s.out2[v] = 1;
	}
	// Add new in2
	for (int i = revg.head[b]; ~i; i = revg.edge[i].next)
	{
		int v = revg.edge[i].v;
		if (s.core2[v] == -1)
			s.in2[v] = 1;
	}

	// Add to s	
	s.first[s.TAM] = a;
	s.second[s.TAM] = b;
	s.TAM++;
}

__device__
bool CheckPrev(const State &s, int a, int b, VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(vetAux.m1, vetAux.pred1, tmp, vetAux.sizeM1, vetAux.sizePred1);

	for (int i = 0; i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizePred2 && !flag;j++)
			if (s.core1[tmp[i]] == vetAux.pred2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.m2, vetAux.pred2, tmp, vetAux.sizeM1, vetAux.sizePred2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizePred1 && !flag;j++)
			if (s.core2[tmp[i]] == vetAux.pred1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckSucc(const State &s, int a, int b, VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(vetAux.m1, vetAux.succ1, tmp, vetAux.sizeM1, vetAux.sizeSucc1);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0; j < vetAux.sizeSucc2 && !flag;j++)
			if (s.core1[tmp[i]] == vetAux.succ2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.m2, vetAux.succ2, tmp, vetAux.sizeM2, vetAux.sizeSucc2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizeSucc1 && !flag;j++)
			if (s.core2[tmp[i]] == vetAux.succ1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckIn(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.succ1, vetAux.tin1, tmp, vetAux.sizeSucc1, vetAux.sizeTin1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.succ2, vetAux.tin2, tmp, vetAux.sizeSucc2, vetAux.sizeTin2);

	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.pred1, vetAux.tin1, tmp, vetAux.sizePred1, vetAux.sizeTin1);

	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred2, vetAux.tin2, tmp, vetAux.sizePred2, vetAux.sizeTin2);

	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckOut(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.succ1, vetAux.tout1, tmp, vetAux.sizeSucc1, vetAux.sizeTout1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.succ2, vetAux.tout2, tmp, vetAux.sizeSucc2, vetAux.sizeTout2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred1, vetAux.tout1, tmp, vetAux.sizePred1, vetAux.sizeTout1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred2, vetAux.tout2, tmp, vetAux.sizePred2, vetAux.sizeTout2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckNew(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.ns1, vetAux.pred1, tmp, vetAux.sizeNS1, vetAux.sizePred1);
	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns2, vetAux.pred2, tmp, vetAux.sizeNS2, vetAux.sizePred2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns1, vetAux.succ1, tmp, vetAux.sizeNS1, vetAux.sizeSucc1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns2, vetAux.succ2, tmp, vetAux.sizeNS2, vetAux.sizeSucc2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}

__device__
void CalCheckVec(int a, int b, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Init
	vetAux.sizePred1 = 0, vetAux.sizePred2 = 0, vetAux.sizeSucc1 = 0, vetAux.sizeSucc2 = 0;

	// aPred
	for (int i = revpat.head[a]; ~i; i = revpat.edge[i].next)
		vetAux.pred1[vetAux.sizePred1++] = revpat.edge[i].v;

	// bPred
	for (int i = revg.head[b]; ~i; i = revg.edge[i].next)
		vetAux.pred2[vetAux.sizePred2++] = revg.edge[i].v;

	// aSucc
	for (int i = pat.head[a]; ~i; i = pat.edge[i].next)
		vetAux.succ1[vetAux.sizeSucc1++] = pat.edge[i].v;

	// bSucc
	for (int i = g.head[b]; ~i; i = g.edge[i].next)
		vetAux.succ2[vetAux.sizeSucc2++] = g.edge[i].v;

	// Sort
	if (vetAux.sizePred1 > 0) quicksort(vetAux.pred1, 0, vetAux.sizePred1 - 1);
	if (vetAux.sizePred2 > 0) quicksort(vetAux.pred2, 0, vetAux.sizePred2 - 1);
	if (vetAux.sizeSucc1 > 0) quicksort(vetAux.succ1, 0, vetAux.sizeSucc1 - 1);
	if (vetAux.sizeSucc2 > 0) quicksort(vetAux.succ2, 0, vetAux.sizeSucc2 - 1);
}

__device__
bool dfs(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	int allPairsFirst[maxe], allPairsSecond[maxe];
	int candiPairsFirst[maxe], candiPairsSecond[maxe];
	
	// Matched
	if ((int)s.TAM == pat.vn)
	{		
		if (FinalCheck(s, pat, g))
		{
			return 1;
		}		
	}

	// Generate Pair(n,m)
	int sizeAllPairs = GenPairs(s, allPairsFirst, allPairsSecond, vetAux, pat, g);
	// Check allPairs, get candiPairs
	int sizeCandiPairs = CheckPairs(s, allPairsFirst, allPairsSecond, candiPairsFirst, candiPairsSecond, sizeAllPairs, vetAux, pat, g, revpat, revg);

	// For tmp dfs store
	int vecFirst[999], vecSecond[999];
	int sizeVec = sizeCandiPairs;
	int m1t[maxv], m2t[maxv];
	int tin1t[maxv], tin2t[maxv];
	int tout1t[maxv], tout2t[maxv];
	int n1t[maxv], n2t[maxv];
	int ns1t[maxv], ns2t[maxv];
	int t1t[maxv], t2t[maxv];

	/*vecFirst = (int*)malloc(sizeCandiPairs * sizeof(int));
	vecSecond = (int*)malloc(sizeCandiPairs * sizeof(int));*/
	memcpy(vecFirst, candiPairsFirst, sizeCandiPairs * sizeof(int));
	memcpy(vecSecond, candiPairsSecond, sizeCandiPairs * sizeof(int));

	bool ret = false;
	for (int i = 0;i < sizeVec;i++)
	{
		State ns = s;

		int a = vecFirst[i], b = vecSecond[i];
		UpdateState(ns, a, b, pat, g, revpat, revg);

		memcpy(m1t, vetAux.m1, maxv * sizeof(int));
		memcpy(m2t, vetAux.m2, maxv * sizeof(int));

		memcpy(tin1t, vetAux.tin1, maxv * sizeof(int));
		memcpy(tin2t, vetAux.tin2, maxv * sizeof(int));

		memcpy(tout1t, vetAux.tout1, maxv * sizeof(int));
		memcpy(tout2t, vetAux.tout2, maxv * sizeof(int));

		memcpy(n1t, vetAux.n1, maxv * sizeof(int));
		memcpy(n2t, vetAux.n2, maxv * sizeof(int));

		memcpy(ns1t, vetAux.ns1, maxv * sizeof(int));
		memcpy(ns2t, vetAux.ns2, maxv * sizeof(int));

		memcpy(t1t, vetAux.t1, maxv * sizeof(int));
		memcpy(t2t, vetAux.t2, maxv * sizeof(int));
		
		ret = dfs(ns, vetAux, pat, g, revpat, revg);		

		memcpy(vetAux.m1, m1t, maxv * sizeof(int));
		memcpy(vetAux.m2, m2t, maxv * sizeof(int));

		memcpy(vetAux.tin1, tin1t, maxv * sizeof(int));
		memcpy(vetAux.tin2, tin2t, maxv * sizeof(int));

		memcpy(vetAux.tout1, tout1t, maxv * sizeof(int));
		memcpy(vetAux.tout2, tout2t, maxv * sizeof(int));

		memcpy(vetAux.n1, n1t, maxv * sizeof(int));
		memcpy(vetAux.n2, n2t, maxv * sizeof(int));

		memcpy(vetAux.ns1, ns1t, maxv * sizeof(int));
		memcpy(vetAux.ns2, ns2t, maxv * sizeof(int));

		memcpy(vetAux.t1, t1t, maxv * sizeof(int));
		memcpy(vetAux.t2, t2t, maxv * sizeof(int));

		if (ret) break;
	}

	/*free(allPairsFirst);
	free(allPairsSecond);
	free(candiPairsFirst);
	free(candiPairsSecond);
	free(vecFirst);
	free(vecSecond);*/

	if (ret)
		return 1;
	else return 0;
}

__device__
bool query(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	return dfs(s, vetAux, pat, g, revpat, revg);	
}

//As discussed in detail in Multiprocessor Level, the fewer registers a kernel uses, the more threads and thread blocks are likely to reside
//on a multiprocessor, which can improve performance.
//Therefore, the compiler uses heuristics to minimize register usage while keeping register spilling and instruction count to a minimum.
//An application can optionally aid these heuristics by providing additional information to the compiler in the form of launch bounds that are 
//specified using the __launch_bounds__() qualifier in the definition of a __global__ function :
__global__ void 
__launch_bounds__(maxThreadsPerBlock, minBlocksPerMultiprocessor)
solve(int NBLOCKS, int NTHREADS, Graph *QueryGraph, Graph *DBGraph, int sizeQuery, int sizeDB, unsigned int *dev_matches)
{
	
	memset(controle, 0, MAX * sizeof(int));
	
	/*printf(" QueryGraph \n");
	printGraph(QueryGraph, sizeQuery);
	printf("\n\n\n DBGraph \n\n\n");
	printGraph(DBGraph, sizeDB);*/

	int init = threadIdx.x + blockIdx.x * blockDim.x;
		
	while (controle[init] < sizeQuery) {
		int j = controle[init];

		if (init >= sizeDB)
			continue;

		Graph pat, g, revpat, revg;
		State s;
		s.init();

		VetAuxiliares vetAux;
		Vertex vtxPat[maxv], vtxRevPat[maxv];
		Edge edgePat[maxe], edgeRevPat[maxe];
		int headPat[maxe], headRevPat[maxe];

		pat = Graph();		
		pat.vtx  = vtxPat;
		pat.edge = edgePat;
		pat.head = headPat;
		pat.init();

		pat.en = QueryGraph[j].en;
		pat.vn = QueryGraph[j].vn;

		for (int k = 0; k < QueryGraph[j].en;k++) {
			pat.edge[k] = QueryGraph[j].edge[k];
			pat.head[k] = QueryGraph[j].head[k];
		}

		for (int k = 0; k < QueryGraph[j].vn;k++) {
			pat.vtx[k] = QueryGraph[j].vtx[k];
		}
		
		revpat = Graph();		
		revpat.vtx = vtxRevPat;
		revpat.edge = edgeRevPat;
		revpat.head = headRevPat;
		revpat.init();

		GenRevGraph(pat, revpat);

		for (int x = init; x < sizeDB; x += NTHREADS * NBLOCKS)
		{
			if (pat.vn > DBGraph[x].vn || pat.en > DBGraph[x].en) continue;

			//printf("x => %d \n", x);

			g = Graph(), revg = Graph();
			Vertex vtxG[maxv], vtxRevG[maxv];
			Edge edgeG[maxe], edgeRevG[maxe];
			int headG[maxe], headRevg[maxe];

			g.vtx = vtxG;
			g.edge = edgeG;
			g.head = headG;
			g.init();
			initGraph(DBGraph[x], g);

			//printf("x => %d pat.vn %d g.vn %d pat.en %d g.en %d \n", x, pat.vn, g.vn, pat.en, g.en);
						
			revg.vtx  = vtxRevG;
			revg.edge = edgeRevG;
			revg.head = headRevg;
			revg.init();
			GenRevGraph(g, revg);
			
			if (query(s, vetAux, pat, g, revpat, revg)) // Matched
			{
				atomicAdd(&dev_matches[j], 1);
			}
		}
		controle[init]++;
	}
	
}

void cudaShowLimit() {
	size_t limit = 0;

	if (hipDeviceGetLimit(&limit, hipLimitStackSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do stack\n");
	}
	//printf("hipLimitStackSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do FIFO\n");
	}
	//printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitMallocHeapSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do HEAP\n");
	}	
	//printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

	limit = 1024 * 128;

	hipDeviceSetLimit(hipLimitStackSize, limit);	

	limit = 1024 * 1024 * 32;

	//hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);

	limit = 1024 * 1024 * 32;

	//hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

	limit = 0;

	hipDeviceGetLimit(&limit, hipLimitStackSize);
	//printf("New hipLimitStackSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
	//printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
	//printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);
}

void beforeSolve() {
	Graph *DBGraphCUDA, *QueryGraphCUDA;
	unsigned int *MatchesCUDA;
	hipError_t cudaStatus;	
	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudaShowLimit();

	QueryGraphCUDA = alocaGraph(QueryGraph, QueryGraphSize);
	DBGraphCUDA = alocaGraph(DBGraph, DBGraphSize);
	
	int sizeofGrafo = DBGraphSize * (sizeof(Graph) + (maxv * sizeof(Vertex)) + (2 * maxe * sizeof(Edge)));
	sizeofGrafo+= QueryGraphSize * (sizeof(Graph) + (maxv * sizeof(Vertex)) + (2 * maxe * sizeof(Edge)));

	//printf("CUDA mem. usage => %d \n", sizeofGrafo);

	hipMalloc((void **)&MatchesCUDA, MAX_GRAPHS_QUERY * sizeof(int));
	cudaStatus = hipMemcpy(MatchesCUDA, matches, (sizeof(int) * MAX_GRAPHS_QUERY), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatchesCUDA h-> d hipMemcpy failed!");
		goto Error;
	}

	printf("Processando...\nBlocks %d Threads %d Modelos %d Grafos %d \n", NBLOCKS, NTHREADS, DBGraphSize, QueryGraphSize);

	solve << <NBLOCKS, NTHREADS >> > (NBLOCKS, NTHREADS, QueryGraphCUDA, DBGraphCUDA, QueryGraphSize, DBGraphSize, MatchesCUDA);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(matches, MatchesCUDA, MAX_GRAPHS_QUERY * sizeof(int), hipMemcpyDeviceToHost);
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatchesCUDA d->h hipMemcpy failed!");
		goto Error;
	}

	
	for (int i = 0; i < QueryGraphSize;i++) {
		printf("%s %d Matches found %d \n", queryPath, i, matches[i]);
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time elapsed %.2f \n", time);	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(QueryGraphCUDA);
	hipFree(DBGraphCUDA);
}

int main(int argc, char* argv[])
{
	if (argc == 3) NBLOCKS = atoi(argv[1]), NTHREADS = atoi(argv[2]);
	if (argc == 4) NBLOCKS = atoi(argv[1]), NTHREADS = atoi(argv[2]), queryPath = argv[3];		
	if (argc == 5) NBLOCKS = atoi(argv[1]), NTHREADS = atoi(argv[2]), queryPath = argv[3], dbPath = argv[4];

	init();
	input();	
	beforeSolve();	
}

