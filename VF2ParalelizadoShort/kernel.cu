#include "hip/hip_runtime.h"
const static int maxv = 10;
const static int maxe = 22;
const int MAX_GRAPHS_DB = 196;
const int MAX_GRAPHS_QUERY = 20;
const int NBLOCKS = 5, NTHREADS = 20;

#include "head.h"
#include "class.h"
#include "signature.h"

#define swap(A,B) { float temp = A; A = B; B = temp;}

const char *QueryPath[MAX_GRAPHS_QUERY]; // Query file path vector
int QueryPathPointer[MAX_GRAPHS_QUERY];
int DBGraphSize, QueryGraphSize, QueryPathSize;

Graph DBGraph[MAX_GRAPHS_DB], QueryGraph[MAX_GRAPHS_QUERY], *vec;

__device__
int contador = 0;
unsigned int matches[MAX_GRAPHS_QUERY];

void init()
{
	ofstream fout;
	fout.open("time.txt");
	fout.close();
	memset(matches, 0, MAX_GRAPHS_QUERY * sizeof(int));
}

string dataset() {
	string dbPath = "Data/db/Q10.min.data";
	QueryPathSize = 0;

	QueryPath[QueryPathSize] = "Data/query/Q4.min.my";
	QueryPathPointer[QueryPathSize] = strlen(QueryPath[QueryPathSize]);
	QueryPathSize++;

	return dbPath;
}

void ReadFile(string path, int &graphSize, int MAX_GRAPHS)
{
	bool eof = false;
	graphSize = 0;

	ifstream fin;
	fin.open(path.c_str());

	vec = (Graph*)malloc(MAX_GRAPHS * sizeof(Graph));
	vec[graphSize].aloca();

	string buff;
	int n = -1;
	int m, l;
	int p, q;
	while (getline(fin, buff))
	{

		if (buff.size() == 0) continue;
		if (buff == "t # -1")
		{
			eof = true;
			graphSize++;			
			break;
		}
		if (buff[0] == 't')
		{
			n++;
			if (n == 0) continue;

			graphSize++;
			vec[graphSize].aloca();
		}
		else if (buff[0] == 'v')
		{
			sscanf_s(buff.c_str(), "v %d %d", &m, &l);
			vec[graphSize].addv(m, l);
		}
		else if (buff[0] == 'e')
		{
			sscanf_s(buff.c_str(), "e %d %d %d", &p, &q, &l);
			vec[graphSize].adde(p, q, l);

		}
		else puts("Error!");
	}

	if (!eof)
		printf("Nao foi encontrado o fim do arquivo (t #-1) \n");

	fin.close();
}

void ReadDB(string path)
{
	ReadFile(path, DBGraphSize, MAX_GRAPHS_DB);
	
	for (int i = 0; i < DBGraphSize;i++) {
		DBGraph[i].en = vec[i].en;
		DBGraph[i].vn = vec[i].vn;

		DBGraph[i].vtx  = vec[i].vtx;
		DBGraph[i].edge = vec[i].edge;
		DBGraph[i].head = vec[i].head;
	}
}

void ReadQuery(string path)
{
	ReadFile(path, QueryGraphSize, MAX_GRAPHS_QUERY);

	for (int i = 0; i < QueryGraphSize;i++) {
		QueryGraph[i].en = vec[i].en;
		QueryGraph[i].vn = vec[i].vn;

		QueryGraph[i].vtx  = vec[i].vtx;
		QueryGraph[i].edge = vec[i].edge;
		QueryGraph[i].head = vec[i].head;
	}

}
__device__
void GenRevGraph(const Graph &src, Graph &dst)
{
	dst = Graph();
	dst.aloca();

	for (int i = 0; i < src.vn; i++)
		dst.addv(src.vtx[i].id, src.vtx[i].label);

	for (int i = 0; i < src.en; i++)
		dst.addse(src.edge[i].v, src.edge[i].u, src.edge[i].label);
}

void input()
{
	// Standard data set
	string dbPath = dataset();

	string tt = "Output/ans";
	for (int i = 0;i < (int)QueryPathSize;i++) {
		ReadQuery(QueryPath[i]);
	}

	ReadDB(dbPath);
	puts("Read Data Finished!");
}

char* allocaString(const char **str, int size) {
	char *localCUDA, *a;
	int TAM = 0;

	//aloca
	for (int i = 0;i < QueryPathSize;i++)
		TAM += QueryPathPointer[i];

	a = (char *)malloc(TAM * sizeof(char));

	//flatten
	int subidx = 0;
	for (int i = 0;i < QueryPathSize;i++)
	{
		for (int j = 0; j < QueryPathPointer[i]; j++)
			a[subidx++] = QueryPath[i][j];
	}

	hipMalloc((void **)&localCUDA, TAM * sizeof(char));
	hipMemcpy(localCUDA, a, TAM * sizeof(char), hipMemcpyHostToDevice);

	return localCUDA;
}

Graph* alocaGraph(Graph *Grafo, int GraphSize) {
	Graph *GraphHost, *GraphCUDA;

	GraphHost = (Graph*)malloc(GraphSize * sizeof(Graph));

	for (int k = 0;k < GraphSize;k++) {
		Vertex *vtx;
		Edge *edge;
		int *head;

		if (hipMalloc((void **)&vtx, Grafo[k].vn * sizeof(Vertex)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&edge, Grafo[k].en * sizeof(Edge)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&head, maxe * sizeof(int)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar o head \n");
		}

		if (hipMemcpy(vtx, Grafo[k].vtx, Grafo[k].vn * sizeof(Vertex), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar os vertices \n");
		}

		if (hipMemcpy(edge, Grafo[k].edge, Grafo[k].en * sizeof(Edge), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar as arestas \n");
		}

		if (hipMemcpy(head, Grafo[k].head, maxe * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar o head \n");
		}

		GraphHost[k].vtx = vtx;
		GraphHost[k].edge = edge;
		GraphHost[k].head = head;
		GraphHost[k].en = Grafo[k].en;
		GraphHost[k].vn = Grafo[k].vn;
	}

	hipMalloc((void **)&GraphCUDA, GraphSize * sizeof(Graph));
	hipMemcpy(GraphCUDA, GraphHost, (sizeof(Graph) * GraphSize), hipMemcpyHostToDevice);

	return GraphCUDA;
}


__device__
bool FinalCheck(const State &s, Graph &pat, Graph &g)
{
	for (int i = 0;i < pat.en;i++)
	{
		Edge e1 = pat.edge[i];
		bool flag = 0;

		for (int j = g.head[s.core1[e1.u]];~j;j = g.edge[j].next)
		{
			Edge e2 = g.edge[j];

			if (e1.label == e2.label&&s.core1[e1.v] == e2.v)
			{
				flag = 1;
				break;
			}
		}
		if (!flag) return 0;
	}
	return 1;
}
__device__
void CalDFSVec(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g)
{
	ClearArrays(vetAux);

	for (int i = 0; i < s.TAM;i++) {
		vetAux.m1[vetAux.sizeM1++] = s.first[i];
		vetAux.m2[vetAux.sizeM2++] = s.second[i];
	}

	if (vetAux.sizeM1 > 0 && vetAux.sizeM2 > 0) {
		quicksort(vetAux.m1, 0, vetAux.sizeM1 - 1);
		quicksort(vetAux.m2, 0, vetAux.sizeM2 - 1);
	}

	for (int i = 0; i < pat.vn; i++) {
		if (s.out1[i])
			vetAux.tout1[vetAux.sizeTout1++] = i;
		if (s.in1[i]) {
			vetAux.tin1[vetAux.sizeTin1++] = i;
		}
		vetAux.n1[vetAux.sizeN1++] = i;
	}

	for (int i = 0; i < g.vn; i++) {
		if (s.out2[i])
			vetAux.tout2[vetAux.sizeTout2++] = i;
		if (s.in2[i])
			vetAux.tin2[vetAux.sizeTin2++] = i;
		vetAux.n2[vetAux.sizeN2++] = i;
	}

	vetAux.sizeT1 = Union(vetAux.tin1, vetAux.tout1, vetAux.t1, vetAux.sizeTin1, vetAux.sizeTout1);

	vetAux.sizeT2 = Union(vetAux.tin1, vetAux.tout2, vetAux.t2, vetAux.sizeTin1, vetAux.sizeTout2);

	int tmp[maxv], sizeTmp;

	sizeTmp = Difference(vetAux.n1, vetAux.m1, tmp, vetAux.sizeN1, vetAux.sizeM1);

	vetAux.sizeNS1 = Difference(tmp, vetAux.t1, vetAux.ns1, sizeTmp, vetAux.sizeT1);

	sizeTmp = Difference(vetAux.n2, vetAux.m2, tmp, vetAux.sizeN2, vetAux.sizeM2);

	vetAux.sizeNS2 = Difference(tmp, vetAux.t2, vetAux.ns2, sizeTmp, vetAux.sizeT2);
}

__device__
bool check(const State &s, int a, int b, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Check vertex label
	if (pat.vtx[a].label != g.vtx[b].label) return 0;

	// Check edge label
	CalCheckVec(a, b, vetAux, pat, g, revpat, revg);

	// Feasibility
	if (CheckPrev(s, a, b, vetAux) && CheckSucc(s, a, b, vetAux) && CheckIn(vetAux) && CheckOut(vetAux) && CheckNew(vetAux)) return 1;

	return 0;
}

__device__
int GenPairs(const State &s, int *&allPairsFirst, int *&allPairsSecond, VetAuxiliares &vetAux, Graph &pat, Graph &g)
{
	int sizeAllPairs = 0;

	CalDFSVec(s, vetAux, pat, g);

	if (vetAux.sizeTout1 > 0 && vetAux.sizeTout2 > 0) {
		allPairsFirst = (int*)malloc(vetAux.sizeTout1 * vetAux.sizeTout2 * sizeof(int));
		allPairsSecond = (int*)malloc(vetAux.sizeTout1 * vetAux.sizeTout2 * sizeof(int));
	}

	for (int i = 0; i < (int)vetAux.sizeTout1; i++)
		for (int j = 0; j < (int)vetAux.sizeTout2; j++) {
			allPairsFirst[sizeAllPairs] = vetAux.tout1[i], allPairsSecond[sizeAllPairs++] = vetAux.tout2[j];
		}

	if (sizeAllPairs > 0)
	{
		return sizeAllPairs;
	}

	if (vetAux.sizeTin1 > 0 && vetAux.sizeTin2 > 0) {
		allPairsFirst = (int*)malloc(vetAux.sizeTin1 * vetAux.sizeTin2 * sizeof(int));
		allPairsSecond = (int*)malloc(vetAux.sizeTin1 * vetAux.sizeTin2 * sizeof(int));
	}

	for (int i = 0; i < (int)vetAux.sizeTin1; i++)
		for (int j = 0; j < (int)vetAux.sizeTin2; j++) {
			allPairsFirst[sizeAllPairs] = vetAux.tin1[i], allPairsSecond[sizeAllPairs++] = vetAux.tin2[j];
		}

	if (sizeAllPairs > 0)
	{
		return sizeAllPairs;
	}

	int temp1[maxv], temp2[maxv];
	int sizeTemp1 = 0, sizeTemp2 = 0;

	for (int i = 0; i < pat.vn; i++)
		if (s.core1[i] == -1)
			temp1[sizeTemp1++] = i;
	
	for (int i = 0; i < g.vn; i++)
		if (s.core2[i] == -1)
			temp2[sizeTemp2++] = i;

	allPairsFirst = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));
	allPairsSecond = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));

	for (int i = 0; i < sizeTemp1; i++)
		for (int j = 0; j < sizeTemp2; j++) {
			allPairsFirst[sizeAllPairs] = temp1[i], allPairsSecond[sizeAllPairs++] = temp2[j];
		}

	return sizeAllPairs;	
}
__device__
int CheckPairs(const State &s, int *&allPairsFirst, int *&allPairsSecond, int *&candiPairsFirst, int *&candiPairsSecond, int sizeAllPairs, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	int sizeCandiPairs = 0;

	candiPairsFirst = (int*)malloc(sizeAllPairs * sizeof(int));
	candiPairsSecond = (int*)malloc(sizeAllPairs * sizeof(int));

	for (int i = 0; i < sizeAllPairs; i++) {
		if (check(s, allPairsFirst[i], allPairsSecond[i], vetAux, pat, g, revpat, revg)) {
			candiPairsFirst[sizeCandiPairs] = allPairsFirst[i];
			candiPairsSecond[sizeCandiPairs++] = allPairsSecond[i];
		}
	}

	return sizeCandiPairs;
}
__device__
void UpdateState(State &s, int a, int b, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Update core,in,out
	for (int i = 0; i < pat.vn; i++)
	{
		s.core1[a] = b;
		s.in1[a] = 0;
		s.out1[a] = 0;
	}
	for (int i = 0; i < g.vn; i++)
	{
		s.core2[b] = a;
		s.in2[b] = 0;
		s.out2[b] = 0;
	}

	for (int i = pat.head[a]; ~i; i = pat.edge[i].next)
	{
		int v = pat.edge[i].v;
		if (s.core1[v] == -1)
			s.out1[v] = 1;
	}
	// Add new in1
	for (int i = revpat.head[a]; ~i; i = revpat.edge[i].next)
	{
		int v = revpat.edge[i].v;
		if (s.core1[v] == -1)
			s.in1[v] = 1;
	}
	// Add new out2
	for (int i = g.head[b]; ~i; i = g.edge[i].next)
	{
		int v = g.edge[i].v;
		if (s.core2[v] == -1)
			s.out2[v] = 1;
	}
	// Add new in2
	for (int i = revg.head[b]; ~i; i = revg.edge[i].next)
	{
		int v = revg.edge[i].v;
		if (s.core2[v] == -1)
			s.in2[v] = 1;
	}

	// Add to s	
	s.first[s.TAM] = a;
	s.second[s.TAM] = b;
	s.TAM++;
}

__device__
bool CheckPrev(const State &s, int a, int b, VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(vetAux.m1, vetAux.pred1, tmp, vetAux.sizeM1, vetAux.sizePred1);

	for (int i = 0; i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizePred2 && !flag;j++)
			if (s.core1[tmp[i]] == vetAux.pred2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.m2, vetAux.pred2, tmp, vetAux.sizeM1, vetAux.sizePred2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizePred1 && !flag;j++)
			if (s.core2[tmp[i]] == vetAux.pred1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckSucc(const State &s, int a, int b, VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(vetAux.m1, vetAux.succ1, tmp, vetAux.sizeM1, vetAux.sizeSucc1);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0; j < vetAux.sizeSucc2 && !flag;j++)
			if (s.core1[tmp[i]] == vetAux.succ2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.m2, vetAux.succ2, tmp, vetAux.sizeM2, vetAux.sizeSucc2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < vetAux.sizeSucc1 && !flag;j++)
			if (s.core2[tmp[i]] == vetAux.succ1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckIn(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.succ1, vetAux.tin1, tmp, vetAux.sizeSucc1, vetAux.sizeTin1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.succ2, vetAux.tin2, tmp, vetAux.sizeSucc2, vetAux.sizeTin2);

	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(vetAux.pred1, vetAux.tin1, tmp, vetAux.sizePred1, vetAux.sizeTin1);

	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred2, vetAux.tin2, tmp, vetAux.sizePred2, vetAux.sizeTin2);

	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckOut(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.succ1, vetAux.tout1, tmp, vetAux.sizeSucc1, vetAux.sizeTout1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.succ2, vetAux.tout2, tmp, vetAux.sizeSucc2, vetAux.sizeTout2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred1, vetAux.tout1, tmp, vetAux.sizePred1, vetAux.sizeTout1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.pred2, vetAux.tout2, tmp, vetAux.sizePred2, vetAux.sizeTout2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckNew(VetAuxiliares &vetAux)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(vetAux.ns1, vetAux.pred1, tmp, vetAux.sizeNS1, vetAux.sizePred1);
	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns2, vetAux.pred2, tmp, vetAux.sizeNS2, vetAux.sizePred2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns1, vetAux.succ1, tmp, vetAux.sizeNS1, vetAux.sizeSucc1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(vetAux.ns2, vetAux.succ2, tmp, vetAux.sizeNS2, vetAux.sizeSucc2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}

__device__
void CalCheckVec(int a, int b, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	// Init
	vetAux.sizePred1 = 0, vetAux.sizePred2 = 0, vetAux.sizeSucc1 = 0, vetAux.sizeSucc2 = 0;

	// aPred
	for (int i = revpat.head[a]; ~i; i = revpat.edge[i].next)
		vetAux.pred1[vetAux.sizePred1++] = revpat.edge[i].v;

	// bPred
	for (int i = revg.head[b]; ~i; i = revg.edge[i].next)
		vetAux.pred2[vetAux.sizePred2++] = revg.edge[i].v;

	// aSucc
	for (int i = pat.head[a]; ~i; i = pat.edge[i].next)
		vetAux.succ1[vetAux.sizeSucc1++] = pat.edge[i].v;

	// bSucc
	for (int i = g.head[b]; ~i; i = g.edge[i].next)
		vetAux.succ2[vetAux.sizeSucc2++] = g.edge[i].v;

	// Sort
	if (vetAux.sizePred1 > 0) quicksort(vetAux.pred1, 0, vetAux.sizePred1 - 1);
	if (vetAux.sizePred2 > 0) quicksort(vetAux.pred2, 0, vetAux.sizePred2 - 1);
	if (vetAux.sizeSucc1 > 0) quicksort(vetAux.succ1, 0, vetAux.sizeSucc1 - 1);
	if (vetAux.sizeSucc2 > 0) quicksort(vetAux.succ2, 0, vetAux.sizeSucc2 - 1);
}

__device__
bool dfs(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	int *allPairsFirst, *allPairsSecond;
	int *candiPairsFirst, *candiPairsSecond;
	
	// Matched
	if ((int)s.TAM == pat.vn)
	{		
		if (FinalCheck(s, pat, g))
		{
			return 1;
		}		
	}

	// Generate Pair(n,m)
	int sizeAllPairs = GenPairs(s, allPairsFirst, allPairsSecond, vetAux, pat, g);
	// Check allPairs, get candiPairs
	int sizeCandiPairs = CheckPairs(s, allPairsFirst, allPairsSecond, candiPairsFirst, candiPairsSecond, sizeAllPairs, vetAux, pat, g, revpat, revg);

	// For tmp dfs store
	int *vecFirst, *vecSecond;
	int sizeVec = sizeCandiPairs;
	int m1t[maxv], m2t[maxv];
	int tin1t[maxv], tin2t[maxv];
	int tout1t[maxv], tout2t[maxv];
	int n1t[maxv], n2t[maxv];
	int ns1t[maxv], ns2t[maxv];
	int t1t[maxv], t2t[maxv];

	vecFirst = (int*)malloc(sizeCandiPairs * sizeof(int));
	vecSecond = (int*)malloc(sizeCandiPairs * sizeof(int));

	memcpy(vecFirst, candiPairsFirst, sizeCandiPairs * sizeof(int));
	memcpy(vecSecond, candiPairsSecond, sizeCandiPairs * sizeof(int));

	bool ret = false;
	
	for (int i = 0;i < sizeVec;i++)
	{
		State ns = s;

		int a = vecFirst[i], b = vecSecond[i];
		
		UpdateState(ns, a, b, pat, g, revpat, revg);

		memcpy(m1t, vetAux.m1, maxv * sizeof(int));
		memcpy(m2t, vetAux.m2, maxv * sizeof(int));

		memcpy(tin1t, vetAux.tin1, maxv * sizeof(int));
		memcpy(tin2t, vetAux.tin2, maxv * sizeof(int));

		memcpy(tout1t, vetAux.tout1, maxv * sizeof(int));
		memcpy(tout2t, vetAux.tout2, maxv * sizeof(int));

		memcpy(n1t, vetAux.n1, maxv * sizeof(int));
		memcpy(n2t, vetAux.n2, maxv * sizeof(int));

		memcpy(ns1t, vetAux.ns1, maxv * sizeof(int));
		memcpy(ns2t, vetAux.ns2, maxv * sizeof(int));

		memcpy(t1t, vetAux.t1, maxv * sizeof(int));
		memcpy(t2t, vetAux.t2, maxv * sizeof(int));
		
		ret = dfs(ns, vetAux, pat, g, revpat, revg);		

		memcpy(vetAux.m1, m1t, maxv * sizeof(int));
		memcpy(vetAux.m2, m2t, maxv * sizeof(int));

		memcpy(vetAux.tin1, tin1t, maxv * sizeof(int));
		memcpy(vetAux.tin2, tin2t, maxv * sizeof(int));

		memcpy(vetAux.tout1, tout1t, maxv * sizeof(int));
		memcpy(vetAux.tout2, tout2t, maxv * sizeof(int));

		memcpy(vetAux.n1, n1t, maxv * sizeof(int));
		memcpy(vetAux.n2, n2t, maxv * sizeof(int));

		memcpy(vetAux.ns1, ns1t, maxv * sizeof(int));
		memcpy(vetAux.ns2, ns2t, maxv * sizeof(int));

		memcpy(vetAux.t1, t1t, maxv * sizeof(int));
		memcpy(vetAux.t2, t2t, maxv * sizeof(int));

		if (ret) break;
	}

	free(allPairsFirst);
	free(allPairsSecond);
	free(candiPairsFirst);
	free(candiPairsSecond);
	free(vecFirst);
	free(vecSecond);

	if (ret)
		return 1;
	else return 0;
}

__device__
bool query(const State &s, VetAuxiliares &vetAux, Graph &pat, Graph &g, Graph &revpat, Graph &revg)
{
	return dfs(s, vetAux, pat, g, revpat, revg);
}
__device__
Graph copyGraph(Graph &graphSource) {
	Graph graphDest;
	graphDest.aloca();

	graphSource.en = graphDest.en;
	graphSource.vn = graphDest.vn;

	for (int j = 0; j < graphSource.en;j++) {
		graphDest.edge[j] = graphSource.edge[j];
		graphDest.head[j] = graphSource.head[j];
	}

	for (int j = 0; j < graphSource.vn;j++) {
		graphDest.vtx[j] = graphSource.vtx[j];
	}

	return graphDest;
}


__global__
void solve(Graph *QueryGraph, Graph *DBGraph, char *QueryPath, int *QueryPathPointer, int sizeQuery, int sizeDB, int sizeQueryP, unsigned int *dev_matches)
{
	int x, result;
	int controle[NBLOCKS * NTHREADS];

	memset(controle, 0, NBLOCKS * NTHREADS * sizeof(int));

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printf("Processando...\nThreads %d Blocks %d Modelos %d Grafos %d Arquivos %d\n", NTHREADS, NBLOCKS, sizeDB, sizeQuery, sizeQueryP);
	}

	/*printf(" QueryGraph \n");
	printGraph(QueryGraph, sizeQuery);
	printf("\n\n\n DBGraph \n\n\n");
	printGraph(DBGraph, sizeDB);*/

	for (int i = 0;i < (int)sizeQueryP;i++)
	{
		int init = threadIdx.x + blockIdx.x * blockDim.x;
		
		while(controle[init] < sizeQuery){
			int j = controle[init];

			if (init >= sizeDB)
				continue;

			Graph pat, g, revpat, revg;
			State s;
			s.init();	

			VetAuxiliares vetAux;
			
			pat.aloca();
			pat.en = QueryGraph[j].en;
			pat.vn = QueryGraph[j].vn;

			for (int k = 0; k < QueryGraph[j].en;k++) {
				pat.edge[k] = QueryGraph[j].edge[k];
				pat.head[k] = QueryGraph[j].head[k];
			}

			for (int k = 0; k < QueryGraph[j].vn;k++) {
				pat.vtx[k] = QueryGraph[j].vtx[k];
			}

			GenRevGraph(pat, revpat);
			
			result = 0;
			for (x = init; x < sizeDB; x+= NTHREADS * NBLOCKS)
			{
				g.aloca();
				g.en = DBGraph[x].en;
				g.vn = DBGraph[x].vn;

				for (int k = 0; k < DBGraph[x].en;k++) {
					g.edge[k] = DBGraph[x].edge[k];
					g.head[k] = DBGraph[x].head[k];
				}

				for (int k = 0; k < DBGraph[x].vn;k++) {
					g.vtx[k] = DBGraph[x].vtx[k];
				}

				//printf("x => %d pat.vn %d g.vn %d pat.en %d g.en %d \n", x, pat.vn, g.vn, pat.en, g.en);

				if (pat.vn > g.vn || pat.en > g.en) continue;

				GenRevGraph(g, revg);
				
				if (query(s, vetAux, pat, g, revpat, revg)) // Matched
				{
					atomicAdd(&dev_matches[j], 1);
				}

				free(g.head);
				free(g.vtx);
				free(g.edge);
			}

			controle[init]++;

			free(pat.head);
			free(pat.vtx);
			free(pat.edge);
		}		
	}
}

void cudaShowLimit() {
	size_t limit = 0;

	if (hipDeviceGetLimit(&limit, hipLimitStackSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do stack\n");
	}
	//printf("hipLimitStackSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do FIFO\n");
	}
	//printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitMallocHeapSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do HEAP\n");
	}	
	//printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

	limit = 1024 * 64;

	hipDeviceSetLimit(hipLimitStackSize, limit);	
	//hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);

	limit = 1024 * 1024 * 1024;
	//hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

	limit = 0;

	hipDeviceGetLimit(&limit, hipLimitStackSize);
	//printf("New hipLimitStackSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
	//printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
	//printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);
}

void beforeSolve() {
	Graph *DBGraphCUDA, *QueryGraphCUDA;
	char *QueryPathCUDA;
	int *QueryPathPointerCUDA;
	unsigned int *MatchesCUDA;
	hipError_t cudaStatus;	
	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudaShowLimit();

	QueryGraphCUDA = alocaGraph(QueryGraph, QueryGraphSize);
	DBGraphCUDA = alocaGraph(DBGraph, DBGraphSize);
	QueryPathCUDA = allocaString(QueryPath, QueryPathSize);

	hipMalloc((void **)&QueryPathPointerCUDA, MAX_GRAPHS_QUERY * sizeof(int));
	cudaStatus = hipMemcpy(QueryPathPointerCUDA, QueryPathPointer, (sizeof(int) * MAX_GRAPHS_QUERY), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "QueryPathPointerCUDA h->d hipMemcpy failed!");
		goto Error;
	}

	hipMalloc((void **)&MatchesCUDA, MAX_GRAPHS_QUERY * sizeof(int));
	cudaStatus = hipMemcpy(MatchesCUDA, matches, (sizeof(int) * MAX_GRAPHS_QUERY), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatchesCUDA h-> d hipMemcpy failed!");
		goto Error;
	}

	solve << <NBLOCKS, NTHREADS >> > (QueryGraphCUDA, DBGraphCUDA, QueryPathCUDA, QueryPathPointerCUDA, QueryGraphSize, DBGraphSize, QueryPathSize, MatchesCUDA);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(matches, MatchesCUDA, MAX_GRAPHS_QUERY * sizeof(int), hipMemcpyDeviceToHost);
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatchesCUDA d->h hipMemcpy failed!");
		goto Error;
	}

	for(int j=0; j < QueryPathSize;j++)
		for (int i = 0; i < QueryGraphSize;i++) {
			printf("%s %d Matches found %d \n", QueryPath[j], i, matches[i]);
		}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time elapsed %.2f \n", time);	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(QueryGraphCUDA);
	hipFree(DBGraphCUDA);
	hipFree(QueryPathCUDA);
}

int main()
{
	// 0: no output matching ans, 1: output matching ans
	init();
	input();
	beforeSolve();	
}

__device__
void printGraph(Graph grafo[], int size) {
	for (int i = 0;i < size; i++) {
		printf("Indice %d Graph[i].en %d Graph[i].vn %d => \n",i, grafo[i].en, grafo[i].vn);

		for (int j = 0; j < grafo[i].en;j++) {
			printf("indice %d Edge[j].u %d Edge[j].v %d Edge[j].next %d\n",j, grafo[i].edge[j].u, grafo[i].edge[j].v, grafo[i].edge[j].next);
		}
		for (int j = 0; j < grafo[i].vn;j++) {
			printf("indice %d Vtx[j].id %d Vtx[j].label %d \n",j, grafo[i].vtx[j].id, grafo[i].vtx[j].label);
		}
	}
}
__device__
void ClearArrays(VetAuxiliares &vetAux) {
	for (int i = 0; i < maxv;i++) {
		vetAux.m1[i] = 0, vetAux.m2[i] = 0;
		vetAux.tin1[i] = 0, vetAux.tin2[i] = 0;
		vetAux.tout1[i] = 0, vetAux.tout2[i] = 0;
		vetAux.n1[i] = 0, vetAux.n2[i] = 0;
		vetAux.ns1[i] = 0, vetAux.ns2[i] = 0;
		vetAux.t1[i] = 0, vetAux.t2[i] = 0;
	}

	vetAux.sizeM1 = 0, vetAux.sizeM2 = 0;
	vetAux.sizeTin1 = 0, vetAux.sizeTin2 = 0;
	vetAux.sizeTout1 = 0, vetAux.sizeTout2 = 0;
	vetAux.sizeN1 = 0, vetAux.sizeN2 = 0;
	vetAux.sizeNS1 = 0, vetAux.sizeNS2 = 0;
}

__device__
void quicksort(int ls[], int l, int r) {
	int i, j, k, p, q;
	int v;
	if (r <= l)
		return;
	v = ls[r];
	i = l - 1;
	j = r;
	p = l - 1;
	q = r;
	for (;;) {
		while (ls[++i] < v);
		while (v < ls[--j])
			if (j == l)
				break;
		if (i >= j)
			break;
		swap(ls[i], ls[j]);
		if (ls[i] == v) {
			p++;
			swap(ls[p], ls[i]);
		}
		if (v == ls[j]) {
			q--;
			swap(ls[q], ls[j]);
		}
	}
	swap(ls[i], ls[r]);
	j = i - 1;
	i++;
	for (k = l; k < p; k++, j--)
		swap(ls[k], ls[j]);
	for (k = r - 1; k > q; k--, i++)
		swap(ls[k], ls[i]);

	quicksort(ls, l, j);
	quicksort(ls, i, r);
}
__device__
int Union(int arr1[], int arr2[], int arr3[], int m, int n)
{
	int i = 0, j = 0, x = 0;

	while (i < m && j < n) {
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
		}
		else
			if (arr2[j] < arr1[i]) {
				arr3[x++] = arr2[j++];
			}
			else {
				arr3[x++] = arr2[j++];
				i++;
			}
	}

	/* Print remaining elements of the larger array */
	while (i < m)
		arr3[x++] = arr1[i++];
	while (j < n)
		arr3[x++] = arr2[j++];

	return x;
}
__device__
int Difference(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, print smaller 
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
			k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++;
				k++;
			}
		// Skip common element 
			else {
				i++;
				j++;
			}
	}

	// printing remaining elements 
	while (i < n1) {

		arr3[x++] = arr1[i++];
		k++;
	}
	while (j < n2) {
		arr2[x++] = arr1[j++];
		k++;
	}

	return x;
}
__device__
int Intersection(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, jump
		if (arr1[i] < arr2[j]) {
			i++, k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++, k++;
			}
			else {
				arr3[x++] = arr1[i++];
				j++;
			}
	}

	return x;
}