#include "hip/hip_runtime.h"
const static int maxv = 10;
const static int maxe = 20;
const int MAX_GRAPHS_DB = 1;
const int MAX_GRAPHS_QUERY = 4;
const int NBLOCKS = 1, NTHREADS = 2;

#include "head.h"
#include "class.h"
#include "signature.h"

#define swap(A,B) { float temp = A; A = B; B = temp;}


const char *QueryPath[MAX_GRAPHS_QUERY]; // Query file path vector
int QueryPathPointer[MAX_GRAPHS_QUERY];
int DBGraphSize, QueryGraphSize, QueryPathSize;

Graph DBGraph[MAX_GRAPHS_DB], QueryGraph[MAX_GRAPHS_QUERY], *vec;

__device__
int pred1[maxv], pred2[maxv],succ1[maxv], succ2[maxv],m1[maxv], m2[maxv], tin1[maxv], tin2[maxv];
__device__
int tout1[maxv], tout2[maxv],n1[maxv], n2[maxv], ns1[maxv], ns2[maxv], t1[maxv], t2[maxv];
__device__
int sizeM1 = 0, sizeM2 = 0, sizeN1 = 0, sizeN2 = 0, sizeNS1 = 0, sizeNS2 = 0, sizeT1 = 0, sizeT2 = 0, sizeTout1 = 0, sizeTout2 = 0;
__device__
int sizePred1 = 0, sizePred2 = 0, sizeSucc1 = 0, sizeSucc2 = 0, sizeTin1 = 0, sizeTin2 = 0;;
__device__
int sizeAllPairs, sizeCandiPairs;

__device__
Graph pat[NTHREADS], g[NTHREADS], revpat[NTHREADS], revg[NTHREADS];
__device__
int contador = 0;


void init()
{
	ofstream fout;
	fout.open("time.txt");
	fout.close();
}

string dataset() {
	string dbPath = "Data/Q10e10.min.data";
	QueryPathSize = 0;

	QueryPath[QueryPathSize] = "Data/Q4.min.my";
	QueryPathPointer[QueryPathSize] = strlen(QueryPath[QueryPathSize]);
	QueryPathSize++;

	return dbPath;
}

void ReadFile(string path, int &graphSize, int MAX_GRAPHS)
{
	bool eof = false;
	graphSize = 0;

	ifstream fin;
	fin.open(path.c_str());

	vec = (Graph*)malloc(MAX_GRAPHS * sizeof(Graph));
	vec[graphSize].aloca();

	string buff;
	int n;
	int m, l;
	int p, q;
	while (getline(fin, buff))
	{

		if (buff.size() == 0) continue;
		if (buff == "t # -1")
		{
			eof = true;
			graphSize++;			
			break;
		}
		if (buff[0] == 't')
		{
			sscanf_s(buff.c_str(), "t # %d", &n);
			if (n == 0) continue;

			graphSize++;
			vec[graphSize].aloca();
		}
		else if (buff[0] == 'v')
		{
			sscanf_s(buff.c_str(), "v %d %d", &m, &l);
			vec[graphSize].addv(m, l);
		}
		else if (buff[0] == 'e')
		{
			sscanf_s(buff.c_str(), "e %d %d %d", &p, &q, &l);
			vec[graphSize].adde(p, q, l);

		}
		else puts("Error!");
	}

	if (!eof)
		printf("Nao foi encontrado o fim do arquivo (t #-1) \n");

	fin.close();
}

void ReadDB(string path)
{
	ReadFile(path, DBGraphSize, MAX_GRAPHS_DB);

	for (int i = 0; i < DBGraphSize;i++) {
		DBGraph[i].en = vec[i].en;
		DBGraph[i].vn = vec[i].vn;

		DBGraph[i].vtx = vec[i].vtx;
		DBGraph[i].edge = vec[i].edge;
		DBGraph[i].head = vec[i].head;
	}
}

void ReadQuery(string path)
{
	ReadFile(path, QueryGraphSize, MAX_GRAPHS_QUERY);

	for (int i = 0; i < QueryGraphSize;i++) {
		QueryGraph[i].en = vec[i].en;
		QueryGraph[i].vn = vec[i].vn;

		QueryGraph[i].vtx = vec[i].vtx;
		QueryGraph[i].edge = vec[i].edge;
		QueryGraph[i].head = vec[i].head;

	}

}
__device__
void GenRevGraph(const Graph &src, Graph &dst)
{
	dst = Graph();
	dst.aloca();

	for (int i = 0; i < src.vn; i++)
		dst.addv(src.vtx[i].id, src.vtx[i].label);

	for (int i = 0; i < src.en; i++)
		dst.addse(src.edge[i].v, src.edge[i].u, src.edge[i].label);
}

void input()
{
	// Standard data set
	string dbPath = dataset();

	string tt = "Output/ans";
	for (int i = 0;i < (int)QueryPathSize;i++) {
		ReadQuery(QueryPath[i]);
	}

	ReadDB(dbPath);
	puts("Read Data Finished!");
}

char* allocaString(const char **str, int size) {
	char *localCUDA, *a;
	int TAM = 0;

	//aloca
	for (int i = 0;i < QueryPathSize;i++)
		TAM += QueryPathPointer[i];

	a = (char *)malloc(TAM * sizeof(char));

	//flatten
	int subidx = 0;
	for (int i = 0;i < QueryPathSize;i++)
	{
		for (int j = 0; j < QueryPathPointer[i]; j++)
			a[subidx++] = QueryPath[i][j];
	}

	hipMalloc((void **)&localCUDA, TAM * sizeof(char));
	hipMemcpy(localCUDA, a, TAM * sizeof(char), hipMemcpyHostToDevice);

	return localCUDA;
}

Graph* alocaGraph(Graph Grafo[MAX_GRAPHS_DB], int GraphSize) {
	Graph *GraphHost, *GraphCUDA;

	GraphHost = (Graph*)malloc(GraphSize * sizeof(Graph));

	for (int k = 0;k < GraphSize;k++) {
		Vertex *vtx;
		Edge *edge;
		int *head;

		if (hipMalloc((void **)&vtx, Grafo[k].vn * sizeof(Vertex)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&edge, Grafo[k].en * sizeof(Edge)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar os vertices \n");
		}
		if (hipMalloc((void **)&head, maxe * sizeof(int)) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel alocar o head \n");
		}

		if (hipMemcpy(vtx, Grafo[k].vtx, Grafo[k].vn * sizeof(Vertex), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar os vertices \n");
		}

		if (hipMemcpy(edge, Grafo[k].edge, Grafo[k].en * sizeof(Edge), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar as arestas \n");
		}

		if (hipMemcpy(head, Grafo[k].head, maxe * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
			printf("ERROR: N�o foi poss�vel copiar o head \n");
		}

		GraphHost[k].vtx = vtx;
		GraphHost[k].edge = edge;
		GraphHost[k].head = head;
		GraphHost[k].en = Grafo[k].en;
		GraphHost[k].vn = Grafo[k].vn;
	}

	hipMalloc((void **)&GraphCUDA, GraphSize * sizeof(Graph));
	hipMemcpy(GraphCUDA, GraphHost, (sizeof(Graph) * GraphSize), hipMemcpyHostToDevice);

	return GraphCUDA;
}

__device__
bool FinalCheck(const State &s, const int threadId)
{
	for (int i = 0;i < pat[threadId].en;i++)
	{
		Edge e1 = pat[threadId].edge[i];
		bool flag = 0;

		for (int j = g[threadId].head[s.core1[e1.u]];~j;j = g[threadId].edge[j].next)
		{
			Edge e2 = g[threadId].edge[j];

			if (e1.label == e2.label&&s.core1[e1.v] == e2.v)
			{
				flag = 1;
				break;
			}
		}
		if (!flag) return 0;
	}
	return 1;
}
__device__
void CalDFSVec(const State &s, const int threadId)
{
	//printf("CalDFSVec %d \n", s.TAM);
	ClearArrays();

	for (int i = 0; i < s.TAM;i++) {
		m1[sizeM1++] = s.first[i];
		m2[sizeM2++] = s.second[i];
	}

	//printf("CalDFSVec antes quicksort %d %d\n", sizeM1, sizeM2);

	if (sizeM1 > 0 && sizeM2 > 0) {
		quicksort(m1, 0, sizeM1 - 1);
		quicksort(m2, 0, sizeM2 - 1);
	}

	//printf("CalDFSVec antes pat.vn %d\n", pat.vn);

	for (int i = 0; i < pat[threadId].vn; i++) {
		if (s.out1[i])
			tout1[sizeTout1++] = i;
		if (s.in1[i]) {
			tin1[sizeTin1++] = i;
		}
		n1[sizeN1++] = i;
	}

	//printf("CalDFSVec antes g.vn %d\n", g.vn);

	for (int i = 0; i < g[threadId].vn; i++) {
		if (s.out2[i])
			tout2[sizeTout2++] = i;
		if (s.in2[i])
			tin2[sizeTin2++] = i;
		n2[sizeN2++] = i;
	}

	//printf("CalDFSVec antes metodos \n");

	sizeT1 = Union(tin1, tout1, t1, sizeTin1, sizeTout1);

	sizeT2 = Union(tin1, tout2, t2, sizeTin1, sizeTout2);

	int tmp[maxv], sizeTmp;

	sizeTmp = Difference(n1, m1, tmp, sizeN1, sizeM1);

	sizeNS1 = Difference(tmp, t1, ns1, sizeTmp, sizeT1);

	sizeTmp = Difference(n2, m2, tmp, sizeN2, sizeM2);

	sizeNS2 = Difference(tmp, t2, ns2, sizeTmp, sizeT2);

	//printf("fim CalDFSVec \n");
}

__device__
bool check(const State &s, int a, int b, const int threadId)
{
	//printf("check \n");
	// Check vertex label
	if (pat[threadId].vtx[a].label != g[threadId].vtx[b].label) return 0;

	// Check edge label
	CalCheckVec(s, a, b, threadId);

	// Feasibility
	if (CheckPrev(s, a, b) && CheckSucc(s, a, b) && CheckIn(s) && CheckOut(s) && CheckNew(s)) return 1;
	return 0;
}

__device__
void GenPairs(const State &s, int *&allPairsFirst, int *&allPairsSecond, const int threadId)
{
	//printf("GenPairs \n");

	CalDFSVec(s, threadId);

	if (sizeTout1 > 0 && sizeTout2 > 0) {
		allPairsFirst = (int*)malloc(sizeTout1 * sizeTout2 * sizeof(int));
		allPairsSecond = (int*)malloc(sizeTout1 * sizeTout2 * sizeof(int));
	}

	for (int i = 0; i < (int)sizeTout1; i++)
		for (int j = 0; j < (int)sizeTout2; j++) {
			allPairsFirst[sizeAllPairs] = tout1[i], allPairsSecond[sizeAllPairs++] = tout2[j];
		}

	//printf("1 -> sizeAllPairs %d \n", sizeAllPairs);
	if (sizeAllPairs > 0)
	{
		return;
	}

	if (sizeTin1 > 0 && sizeTin2 > 0) {
		allPairsFirst = (int*)malloc(sizeTin1 * sizeTin2 * sizeof(int));
		allPairsSecond = (int*)malloc(sizeTin1 * sizeTin2 * sizeof(int));
	}

	for (int i = 0; i < (int)sizeTin1; i++)
		for (int j = 0; j < (int)sizeTin2; j++) {
			allPairsFirst[sizeAllPairs] = tin1[i], allPairsSecond[sizeAllPairs++] = tin2[j];
		}

	//printf("2 -> sizeAllPairs %d \n", sizeAllPairs);
	if (sizeAllPairs > 0)
	{
		return;
	}

	int temp1[maxv], temp2[maxv];
	int sizeTemp1 = 0, sizeTemp2 = 0;

	for (int i = 0; i < pat[threadId].vn; i++)
		if (s.core1[i] == -1)
			temp1[sizeTemp1++] = i;
	
	for (int i = 0; i < g[threadId].vn; i++)
		if (s.core2[i] == -1)
			temp2[sizeTemp2++] = i;

	allPairsFirst = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));
	allPairsSecond = (int*)malloc(sizeTemp1 * sizeTemp2 * sizeof(int));

	for (int i = 0; i < sizeTemp1; i++)
		for (int j = 0; j < sizeTemp2; j++) {
			allPairsFirst[sizeAllPairs] = temp1[i], allPairsSecond[sizeAllPairs++] = temp2[j];
		}

	//printf("fim GenPairs %d \n", sizeAllPairs);
}
__device__
void CheckPairs(const State &s, int *&allPairsFirst, int *&allPairsSecond, int *&candiPairsFirst, int *&candiPairsSecond, const int threadId)
{
	//printf("CheckPairs \n");
	sizeCandiPairs = 0;

	candiPairsFirst = (int*)malloc(sizeAllPairs * sizeof(int));
	candiPairsSecond = (int*)malloc(sizeAllPairs * sizeof(int));

	for (int i = 0; i < sizeAllPairs; i++) {
		if (check(s, allPairsFirst[i], allPairsSecond[i], threadId)) {
			candiPairsFirst[sizeCandiPairs] = allPairsFirst[i];
			candiPairsSecond[sizeCandiPairs++] = allPairsSecond[i];
		}
	}

	//printf("fim CheckPairs %d \n", sizeCandiPairs);
}
__device__
void UpdateState(State &s, int a, int b, const int threadId)
{
		// Update core,in,out
	for (int i = 0; i < pat[threadId].vn; i++)
	{
		s.core1[a] = b;
		s.in1[a] = 0;
		s.out1[a] = 0;
	}
	for (int i = 0; i < g[threadId].vn; i++)
	{
		s.core2[b] = a;
		s.in2[b] = 0;
		s.out2[b] = 0;
	}

	for (int i = pat[threadId].head[a]; ~i; i = pat[threadId].edge[i].next)
	{
		int v = pat[threadId].edge[i].v;
		if (s.core1[v] == -1)
			s.out1[v] = 1;
	}
	// Add new in1
	for (int i = revpat[threadId].head[a]; ~i; i = revpat[threadId].edge[i].next)
	{
		int v = revpat[threadId].edge[i].v;
		if (s.core1[v] == -1)
			s.in1[v] = 1;
	}
	// Add new out2
	for (int i = g[threadId].head[b]; ~i; i = g[threadId].edge[i].next)
	{
		int v = g[threadId].edge[i].v;
		if (s.core2[v] == -1)
			s.out2[v] = 1;
	}
	// Add new in2
	for (int i = revg[threadId].head[b]; ~i; i = revg[threadId].edge[i].next)
	{
		int v = revg[threadId].edge[i].v;
		if (s.core2[v] == -1)
			s.in2[v] = 1;
	}

	// Add to s	
	s.first[s.TAM] = a;
	s.second[s.TAM] = b;
	s.TAM++;
}

__device__
bool CheckPrev(const State &s, int a, int b)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(m1, pred1, tmp, sizeM1, sizePred1);

	for (int i = 0; i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < sizePred2 && !flag;j++)
			if (s.core1[tmp[i]] == pred2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(m2, pred2, tmp, sizeM1, sizePred2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < sizePred1 && !flag;j++)
			if (s.core2[tmp[i]] == pred1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckSucc(const State &s, int a, int b)
{
	int tmp[maxv], sizeTmp;
	bool flag;

	sizeTmp = Intersection(m1, succ1, tmp, sizeM1, sizeSucc1);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0; j < sizeSucc2 && !flag;j++)
			if (s.core1[tmp[i]] == succ2[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(m2, succ2, tmp, sizeM2, sizeSucc2);

	for (int i = 0;i < sizeTmp;i++)
	{
		flag = 0;
		for (int j = 0;j < sizeSucc1 && !flag;j++)
			if (s.core2[tmp[i]] == succ1[j])
			{
				flag = 1;
			}
		if (!flag) return 0;
	}

	return 1;
}
__device__
bool CheckIn(const State &s)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(succ1, tin1, tmp, sizeSucc1, sizeTin1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(succ2, tin2, tmp, sizeSucc2, sizeTin2);

	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;

	sizeTmp = Intersection(pred1, tin1, tmp, sizePred1, sizeTin1);

	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(pred2, tin2, tmp, sizePred2, sizeTin2);

	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckOut(const State &s)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(succ1, tout1, tmp, sizeSucc1, sizeTout1);

	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(succ2, tout2, tmp, sizeSucc2, sizeTout2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(pred1, tout1, tmp, sizePred1, sizeTout1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(pred2, tout2, tmp, sizePred2, sizeTout2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}
__device__
bool CheckNew(const State &s)
{
	int tmp[maxv], sizeTmp;
	int a, b, c, d;

	sizeTmp = Intersection(ns1, pred1, tmp, sizeNS1, sizePred1);
	a = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(ns2, pred2, tmp, sizeNS2, sizePred2);
	b = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(ns1, succ1, tmp, sizeNS1, sizeSucc1);
	c = sizeTmp;

	//clear tmp
	for (int i = 0;i < maxv;i++) tmp[i] = 0;
	sizeTmp = Intersection(ns2, succ2, tmp, sizeNS2, sizeSucc2);
	d = sizeTmp;

	return (a <= b) && (c <= d);
}

__device__
void CalCheckVec(const State &s, int a, int b, const int threadId)
{
	//printf("CalCheckVec \n");
	// Init
	sizePred1 = 0, sizePred2 = 0, sizeSucc1 = 0, sizeSucc2 = 0;

	// aPred
	for (int i = revpat[threadId].head[a]; ~i; i = revpat[threadId].edge[i].next)
		pred1[sizePred1++] = revpat[threadId].edge[i].v;

	// bPred
	for (int i = revg[threadId].head[b]; ~i; i = revg[threadId].edge[i].next)
		pred2[sizePred2++] = revg[threadId].edge[i].v;

	// aSucc
	for (int i = pat[threadId].head[a]; ~i; i = pat[threadId].edge[i].next)
		succ1[sizeSucc1++] = pat[threadId].edge[i].v;

	// bSucc
	for (int i = g[threadId].head[b]; ~i; i = g[threadId].edge[i].next)
		succ2[sizeSucc2++] = g[threadId].edge[i].v;

	//printf("antes sort %d %d %d %d\n", sizePred1, sizePred2, sizeSucc1, sizeSucc2);
	//printf(" antes sort => %d %d \n",sizePred1, pred1[0]);
	// Sort
	if (sizePred1 > 0) quicksort(pred1, 0, sizePred1 - 1);
	//printf("a \n");
	if (sizePred2 > 0) quicksort(pred2, 0, sizePred2 - 1);
	//printf("b \n");
	if (sizeSucc1 > 0) quicksort(succ1, 0, sizeSucc1 - 1);
	//printf("c \n");
	if (sizeSucc2 > 0) quicksort(succ2, 0, sizeSucc2 - 1);

	//printf("fim CalCheckVec \n");
}
__device__
void quicksort(int ls[], int l, int r) {
	int i, j, k, p, q;
	int v;
	if (r <= l)
		return;
	v = ls[r];
	i = l - 1;
	j = r;
	p = l - 1;
	q = r;
	for (;;) {
		while (ls[++i] < v);
		while (v < ls[--j])
			if (j == l)
				break;
		if (i >= j)
			break;
		swap(ls[i], ls[j]);
		if (ls[i] == v) {
			p++;
			swap(ls[p], ls[i]);
		}
		if (v == ls[j]) {
			q--;
			swap(ls[q], ls[j]);
		}
	}
	swap(ls[i], ls[r]);
	j = i - 1;
	i++;
	for (k = l; k < p; k++, j--)
		swap(ls[k], ls[j]);
	for (k = r - 1; k > q; k--, i++)
		swap(ls[k], ls[i]);

	quicksort(ls, l, j);
	quicksort(ls, i, r);
}
__device__
int Union(int arr1[], int arr2[], int arr3[], int m, int n)
{
	int i = 0, j = 0, x = 0;

	while (i < m && j < n) {
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
		}
		else
			if (arr2[j] < arr1[i]) {
				arr3[x++] = arr2[j++];
			}
			else {
				arr3[x++] = arr2[j++];
				i++;
			}
	}

	/* Print remaining elements of the larger array */
	while (i < m)
		arr3[x++] = arr1[i++];
	while (j < n)
		arr3[x++] = arr2[j++];

	return x;
}
__device__
int Difference(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, print smaller 
		if (arr1[i] < arr2[j]) {
			arr3[x++] = arr1[i++];
			k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++;
				k++;
			}
		// Skip common element 
			else {
				i++;
				j++;
			}
	}

	// printing remaining elements 
	while (i < n1) {

		arr3[x++] = arr1[i++];
		k++;
	}
	while (j < n2) {
		arr2[x++] = arr1[j++];
		k++;
	}

	return x;
}
__device__
int Intersection(int arr1[], int arr2[], int arr3[], int n1, int n2)
{
	int i = 0, j = 0, k = 0, x = 0;
	while (i < n1 && j < n2) {

		// If not common, jump
		if (arr1[i] < arr2[j]) {
			i++, k++;
		}
		else
			if (arr2[j] < arr1[i]) {
				j++, k++;
			}
			else {
				arr3[x++] = arr1[i++];
				j++;
			}
	}

	return x;
}
__device__
void ClearArrays() {
	//printf("ClearArrays\n");

	for (int i = 0; i < maxv;i++) {
		m1[i] = 0, m2[i] = 0;
		tin1[i] = 0, tin2[i] = 0;
		tout1[i] = 0, tout2[i] = 0;
		n1[i] = 0, n2[i] = 0;
		ns1[i] = 0, ns2[i] = 0;
		t1[i] = 0, t2[i] = 0;
	}

	sizeM1 = 0, sizeM2 = 0;
	sizeTin1 = 0, sizeTin2 = 0;
	sizeTout1 = 0, sizeTout2 = 0;
	sizeN1 = 0, sizeN2 = 0;
	sizeNS1 = 0, sizeNS2 = 0;
	sizeAllPairs = 0;

	//printf("fim ClearArrays");
}

__device__
bool dfs(const State &s, const int threadId)
{
	int *allPairsFirst, *allPairsSecond;
	int *candiPairsFirst, *candiPairsSecond;

	printf("threadId %d contador %d ref s => %d\n", threadId, contador, &s);

	contador++;

	// Matched
	//printf("s.TAM %d pat.vn %d \n", s.TAM, pat[threadId].vn);
	if ((int)s.TAM == pat[threadId].vn)
	{		
		if (FinalCheck(s, threadId))
		{
			return 1;
		}		
	}

	// Generate Pair(n,m)
	GenPairs(s, allPairsFirst, allPairsSecond, threadId);
	// Check allPairs, get candiPairs
	CheckPairs(s, allPairsFirst, allPairsSecond, candiPairsFirst, candiPairsSecond, threadId);

	// For tmp dfs store
	int *vecFirst, *vecSecond;
	int sizeVec = sizeCandiPairs;
	int m1t[maxv], m2t[maxv];
	int tin1t[maxv], tin2t[maxv];
	int tout1t[maxv], tout2t[maxv];
	int n1t[maxv], n2t[maxv];
	int ns1t[maxv], ns2t[maxv];
	int t1t[maxv], t2t[maxv];

	vecFirst = (int*)malloc(sizeCandiPairs * sizeof(int));
	vecSecond = (int*)malloc(sizeCandiPairs * sizeof(int));

	memcpy(vecFirst, candiPairsFirst, sizeCandiPairs * sizeof(int));
	memcpy(vecSecond, candiPairsSecond, sizeCandiPairs * sizeof(int));

	bool ret = false;
	//printf("threadId %d sizeVec %d \n", threadId, sizeVec);
	// Next recursive	
	for (int i = 0;i < sizeVec;i++)
	{
		State ns = s;
		int a = vecFirst[i], b = vecSecond[i];
		
		UpdateState(ns, a, b, threadId);

		memcpy(m1t, m1, maxv * sizeof(int));
		memcpy(m2t, m2, maxv * sizeof(int));

		memcpy(tin1t, tin1, maxv * sizeof(int));
		memcpy(tin2t, tin2, maxv * sizeof(int));

		memcpy(tout1t, tout1, maxv * sizeof(int));
		memcpy(tout2t, tout2, maxv * sizeof(int));

		memcpy(n1t, n1, maxv * sizeof(int));
		memcpy(n2t, n2, maxv * sizeof(int));

		memcpy(ns1t, ns1, maxv * sizeof(int));
		memcpy(ns2t, ns2, maxv * sizeof(int));

		memcpy(t1t, t1, maxv * sizeof(int));
		memcpy(t2t, t2, maxv * sizeof(int));

		ret = dfs(ns, threadId);

		memcpy(m1, m1t, maxv * sizeof(int));
		memcpy(m2, m2t, maxv * sizeof(int));

		memcpy(tin1, tin1t, maxv * sizeof(int));
		memcpy(tin2, tin2t, maxv * sizeof(int));

		memcpy(tout1, tout1t, maxv * sizeof(int));
		memcpy(tout2, tout2t, maxv * sizeof(int));

		memcpy(n1, n1t, maxv * sizeof(int));
		memcpy(n2, n2t, maxv * sizeof(int));

		memcpy(ns1, ns1t, maxv * sizeof(int));
		memcpy(ns2, ns2t, maxv * sizeof(int));

		memcpy(t1, t1t, maxv * sizeof(int));
		memcpy(t2, t2t, maxv * sizeof(int));

		if (ret) break;
	}

	free(allPairsFirst);
	free(allPairsSecond);
	free(candiPairsFirst);
	free(candiPairsSecond);
	free(vecFirst);
	free(vecSecond);

	if (ret)
		return 1;
	else return 0;
}

__device__
bool query(const int threadId, const State &s)
{
	//printf("Referencia s => %d \n", &s);

	return dfs(s, threadId);
}

__global__
void solve(Graph *QueryGraph, Graph *DBGraph, char *QueryPath, int *QueryPathPointer, int sizeQuery, int sizeDB, int sizeQueryP)
{
	int matches = 0;
	State s[NTHREADS];

	if (threadIdx.x == 0)
		printf("Processando qtde modelos %d qtde grafos %d qtde arquivos %d\n", sizeDB, sizeQuery, sizeQueryP);

	/*printf(" QueryGraph \n");
	printGraph(QueryGraph, sizeQuery);
	printf("\n\n\n DBGraph \n\n\n");
	printGraph(DBGraph, sizeDB);*/

	for (int i = 0;i < (int)sizeQueryP;i++)
	{
		for (int j = threadIdx.x;j < sizeQuery;j += NTHREADS) {
			matches = 0;

			s[threadIdx.x].init();
			

			pat[threadIdx.x] = QueryGraph[j];

			GenRevGraph(pat[threadIdx.x], revpat[threadIdx.x]);

			for (int x = 0; x < sizeDB; x++)
			{
				g[threadIdx.x] = DBGraph[x];

				//printf("pat.vn %d  g.vn %d pat.en %d g.en %d \n", pat.vn, g.vn, pat.en, g.en);
				if (pat[threadIdx.x].vn > g[threadIdx.x].vn || pat[threadIdx.x].en > g[threadIdx.x].en) continue;

				GenRevGraph(g[threadIdx.x], revg[threadIdx.x]);

				if (query(threadIdx.x, s[threadIdx.x])) // Matched
				{
					matches++;
				}
			}
			
			printf("%s %d Matches found %d \n", QueryPath, j , matches);
		}		
	}
}

void cudaShowLimit() {
	size_t limit = 0;

	if (hipDeviceGetLimit(&limit, hipLimitStackSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do stack\n");
	}
	printf("hipLimitStackSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do FIFO\n");
	}
	printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);

	if (hipDeviceGetLimit(&limit, hipLimitMallocHeapSize) != hipSuccess) {
		printf("ERROR: N�o foi poss�vel retornar o limite do HEAP\n");
	}	
	printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

	limit = 1024 * 32;

	hipDeviceSetLimit(hipLimitStackSize, limit);	
	//hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);

	limit = 1024 * 1024 * 1024;
	//hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

	limit = 0;

	hipDeviceGetLimit(&limit, hipLimitStackSize);
	printf("New hipLimitStackSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
	printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
	printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);
}

void beforeSolve() {
	Graph *DBGraphCUDA, *QueryGraphCUDA;
	char *QueryPathCUDA;
	int *QueryPathPointerCUDA;
	
	hipError_t cudaStatus;	
	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudaShowLimit();

	QueryGraphCUDA = alocaGraph(QueryGraph, QueryGraphSize);
	DBGraphCUDA = alocaGraph(DBGraph, DBGraphSize);
	QueryPathCUDA = allocaString(QueryPath, QueryPathSize);

	hipMalloc((void **)&QueryPathPointerCUDA, MAX_GRAPHS_QUERY * sizeof(int));
	hipMemcpy(QueryPathPointerCUDA, QueryPathPointer, (sizeof(int) * MAX_GRAPHS_QUERY), hipMemcpyHostToDevice);

	solve << <NBLOCKS, NTHREADS >> > (QueryGraphCUDA, DBGraphCUDA, QueryPathCUDA, QueryPathPointerCUDA, QueryGraphSize, DBGraphSize, QueryPathSize);
	
	//is used in host code (i.e. running on the CPU) when it is desired that CPU activity wait on the completion of any pending GPU activity
	//hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time elapsed %.2f \n", time);	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(QueryGraphCUDA);
	hipFree(DBGraphCUDA);
	hipFree(QueryPathCUDA);
}

int main()
{
	// 0: no output matching ans, 1: output matching ans
	init();
	input();
	beforeSolve();	
}

__device__
void printGraph(Graph grafo[], int size) {
	for (int i = 0;i < size; i++) {
		printf("Indice %d Graph[i].en %d Graph[i].vn %d => \n",i, grafo[i].en, grafo[i].vn);

		for (int j = 0; j < grafo[i].en;j++) {
			printf("indice %d Edge[j].u %d Edge[j].v %d Edge[j].next %d\n",j, grafo[i].edge[j].u, grafo[i].edge[j].v, grafo[i].edge[j].next);
		}
		for (int j = 0; j < grafo[i].vn;j++) {
			printf("indice %d Vtx[j].id %d Vtx[j].label %d \n",j, grafo[i].vtx[j].id, grafo[i].vtx[j].label);
		}
	}
}